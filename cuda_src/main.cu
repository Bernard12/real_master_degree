#include "matrix/Matrix.cuh"
#include "matrix_utils/svd.cuh"
#include <stdio.h>
#include <chrono>

// CATCH_CUDA_ERR(hipMalloc(&dev_array, sizeof(int) * used_n));
// CATCH_CUDA_ERR(hipMemcpy(dev_array, array, sizeof(int) * used_n, hipMemcpyHostToDevice));

int main() {
//    int i1 = 10, i2 = 10, i3 = 10, i4 = 10, i5 = 10, i6 = 10, i7 = 10;
//    Matrix* mtr = hilbert(i1, i2, i3, i4, i5, i6, i7);
//    Matrix* mtr = hilbert(1000, 1000, 100);

    // Matrix* mtr = hilbert(m,n);
    // Matrix* mtr = hilbert(n,m);
    // int * shapes = new int[3];
    // shapes[0] = n;
    // shapes[1] = m;
    // shapes[2] = k;
    // Matrix* mtr = hilbert(n, m, k);
    // delete[] shapes;
    // auto p = QRDecompositionNaive(mtr);
    // Matrix* res = multiply(p.first, p.second);
    // auto trip = SVDDecomposition(mtr, 10, 1e-6);
    // auto svd = SVDDecompositionwCUB(mtr);
    // show(svd->first, 4, 4);
    // show(svd->second, 4, 4);
    // show(svd->third, 4, 16);
    // printf("HI");
    // auto tmp = multiply(svd->first, svd->second);
    // auto res = multiply(tmp, svd->third);

    // auto tmp = multiply(transpose(svd->third), svd->second);
    // auto res = multiply(tmp, transpose(svd->first));
    // printf("Diff: %f \n", diff(mtr, res));
    // show(res, 4, 16);
    // show(res, 16, 4);
    // printf("Size(%d %d), Diff: %f \n", n, m, diff(mtr, res));
    // Matrix* d_mtr;
    // double* d_mtr_arr;
    // int* d_dims_arr;
    // copyMatrixFromHostToDevice(mtr, &d_mtr, &d_mtr_arr, &d_dims_arr);
    // show(mtr, n, m);
    // int* newShapes = new int[2];
    // newShapes[0] = 8;
    // newShapes[1] = 2;
    // mtr->reshape(newShapes, 2);
    // show(mtr, 8, 2);
//    printf("start tt decomposition %d\n", mtr->shape_length);
//    auto start = chrono::high_resolution_clock::now();
//    auto tt    = TTDecomposition(mtr, 1e-3);
//    auto end   = chrono::high_resolution_clock::now();
//
//    auto diff = chrono::duration_cast<chrono::milliseconds>(end - start);
//    printf("Execution time %f\n", diff.count() / 1000.);

    // show(tt[2], 4, 4);
//    for(auto i : tt) {
//        printf("%d ", i->shape_length);
//    }
//    printf("\n");
//    vector<int> indexes = {0, 1, 0};
//    double val = getValueFromTrain(tt, indexes);
//    printf("Value: %f\n", val);

    // show(res, n, m);
    // show(mtr, n, m, k);
    // tensorTrain(mtr, 1e-6);
    // delete mtr;

    // printf("%d %d %d\n", d_mtr, d_mtr_arr, d_dims_arr);
    // (hipFree(d_mtr));
    // (hipFree(d_mtr_arr));
    // (hipFree(d_dims_arr));
    // delete tmp;
    // delete t;
    // delete res;
    // delete trip->first;
    // delete trip->second;
    // delete trip->third;
    // delete trip;

    double res = 0;
    int r = 8;
    double step = 1. / r;

    auto *cube = sinCube(r, step);

    auto start = chrono::high_resolution_clock::now();
    printf("!%d!\n", cube->total_element_count);
//s
//    for (int i = 0; i < cube->shape_length; i++) {
//        printf("%d\n", cube->real_shape[i]);
//    }

    auto tt = TTDecomposition(cube, 1e-3);
    vector<Matrix *> u;

    for (int i = 0; i < 10; i++) {
        auto us = new Matrix(r, 1);
        for (int j = 0; j < r; j++) {
            us->set(j, 0, step);
        }
        u.push_back(us);
    }

    res = convolution(tt, u);
    auto end = chrono::high_resolution_clock::now();


    printf("Res: %.6f\n", res);

    auto diff = chrono::duration_cast<chrono::milliseconds>(end - start);
    printf("Execution time %f", diff.count() / 1000.);



    return 0;
}
