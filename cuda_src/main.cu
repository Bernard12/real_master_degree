#include "matrix/Matrix.cuh"
#include "matrix_utils/svd.cuh"
#include <stdio.h>

// CATCH_CUDA_ERR(hipMalloc(&dev_array, sizeof(int) * used_n));
// CATCH_CUDA_ERR(hipMemcpy(dev_array, array, sizeof(int) * used_n, hipMemcpyHostToDevice));

int main() {
    int n = 3, m = 3, k = 3;
    Matrix* mtr = hilbert(n, m);
    // int * shapes = new int[3];
    // shapes[0] = n;
    // shapes[1] = m;
    // shapes[2] = k;
    // Matrix* mtr = hilbert(n, m, k);
    // delete[] shapes;
    // auto p = QRDecompositionNaive(mtr);
    // Matrix* res = multiply(p.first, p.second);
    // auto trip = SVDDecomposition(mtr, 10, 1e-6);
    // auto svd = SVDDecompositionwCUB(mtr);
    // show(svd->first, 3, 3);
    // printf("HI");
    // auto tmp = multiply(trip->first, trip->second);
    // auto t = transpose(trip->third);
    // auto res = multiply(tmp, t);
    // printf("Diff: %f \n", diff(mtr, res));
    // printf("Size(%d %d), Diff: %f \n", n, m, diff(mtr, res));
    // Matrix* d_mtr;
    // double* d_mtr_arr;
    // int* d_dims_arr;
    // copyMatrixFromHostToDevice(mtr, &d_mtr, &d_mtr_arr, &d_dims_arr);
    // show(mtr, n, m);
    // int* newShapes = new int[2];
    // newShapes[0] = 8;
    // newShapes[1] = 2;
    // mtr->reshape(newShapes, 2);
    // show(mtr, 8, 2);

    // show(res, n, m);
    // show(mtr, n, m, k);
    // tensorTrain(mtr, 1e-6);
    delete mtr;

    // printf("%d %d %d\n", d_mtr, d_mtr_arr, d_dims_arr);
    // (hipFree(d_mtr));
    // (hipFree(d_mtr_arr));
    // (hipFree(d_dims_arr));
    // delete tmp;
    // delete t;
    // delete res;
    // delete trip->first;
    // delete trip->second;
    // delete trip->third;
    // delete trip;
    return 0;
}
