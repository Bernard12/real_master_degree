#include "matrix/Matrix.hpp"
#include "matrix_utils/svd.hpp"

#define CCE(errValue)                                        \
    do {                                                                \
        if (errValue != hipSuccess) {                                  \
            fprintf(stderr ,"[CUDA-ERROR]-[%s(line:%d)]: %s\n", __FILE__, __LINE__, hipGetErrorString(errValue)); \
            exit(0);                                                    \
        }                                                               \
    } while(0);

// CATCH_CUDA_ERR(hipMalloc(&dev_array, sizeof(int) * used_n));
// CATCH_CUDA_ERR(hipMemcpy(dev_array, array, sizeof(int) * used_n, hipMemcpyHostToDevice));

void copyMatrixFromHostToDevice(Matrix* hostMatrix, Matrix** deviceMatrix) {
    auto host_shape = hostMatrix->shape();
    const int n = host_shape.first, m = host_shape.second;
    Matrix* temp = new Matrix(n, m);

    const int matrix_size = sizeof(double) * n * m;
    CCE(hipMalloc(&temp->matrix, matrix_size));
    CCE(hipMemcpy(temp->matrix, hostMatrix->matrix, sizeof(Matrix) * 1, hipMemcpyHostToDevice));

    CCE(hipMalloc(deviceMatrix, sizeof(Matrix) * 1));
    CCE(hipMemcpy(*deviceMatrix, temp, sizeof(Matrix) * 1, hipMemcpyHostToDevice));

    temp-> matrix = NULL;
    delete temp;

}

int main() {
    Matrix* mtr = new Matrix(5, 5);
    Matrix* dev_m;
    copyMatrixFromHostToDevice(mtr, &dev_m);
    delete mtr;
    show<<<1, 1>>>(dev_m, 5, 5);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize())
    CCE(hipGetLastError());
    return 0;
}
