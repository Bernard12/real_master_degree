#include "matrix/Matrix.hpp"
#include "matrix_utils/operations.cuh"

#define CCE(errValue)                                        \
    do {                                                                \
        if (errValue != hipSuccess) {                                  \
            fprintf(stderr ,"[CUDA-ERROR]-[%s(line:%d)]: %s\n", __FILE__, __LINE__, hipGetErrorString(errValue)); \
            exit(0);                                                    \
        }                                                               \
    } while(0);

// CATCH_CUDA_ERR(hipMalloc(&dev_array, sizeof(int) * used_n));
// CATCH_CUDA_ERR(hipMemcpy(dev_array, array, sizeof(int) * used_n, hipMemcpyHostToDevice));

void copyMatrixFromHostToDevice(Matrix* hostMatrix, Matrix** deviceMatrix) {
    const int n = hostMatrix->n, m = hostMatrix->m;
    Matrix* temp = new Matrix(n, m);

    const int matrix_size = sizeof(double) * n * m;
    CCE(hipMalloc(&temp->matrix, matrix_size));
    CCE(hipMemcpy(temp->matrix, hostMatrix->matrix, sizeof(Matrix) * 1, hipMemcpyHostToDevice));

    CCE(hipMalloc(deviceMatrix, sizeof(Matrix) * 1));
    CCE(hipMemcpy(*deviceMatrix, temp, sizeof(Matrix) * 1, hipMemcpyHostToDevice));

    temp-> matrix = NULL;
    delete temp;

}

int main() {
    Matrix* mtr = new Matrix(5, 5);
    mtr->matrix[0] = 1;
    Matrix* dev_m;
    Matrix* dev_m2;
    copyMatrixFromHostToDevice(mtr, &dev_m);
    copyMatrixFromHostToDevice(mtr, &dev_m2);
    sum<<<16, 32>>>(dev_m, dev_m2);
    show<<<1, 1>>>(dev_m, 5, 5);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize())
    CCE(hipGetLastError());
    delete mtr;
    return 0;
}
