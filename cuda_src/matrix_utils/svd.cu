#include "./svd.cuh"


#define CCE(errValue)                                                   \
    do {                                                                \
        if (errValue != hipSuccess) {                                  \
            fprintf(stderr ,"[CUDA-ERROR]-[%s(line:%d)]: %s\n", __FILE__, __LINE__, hipGetErrorString(errValue)); \
            exit(0);                                                    \
        }                                                               \
    } while(0);

// CATCH_CUDA_ERR(hipMalloc(&dev_array, sizeof(int) * used_n));
// CATCH_CUDA_ERR(hipMemcpy(dev_array, array, sizeof(int) * used_n, hipMemcpyHostToDevice));

void copyMatrixFromHostToDevice(Matrix* hostMatrix, Matrix** deviceMatrix, double** deviceMatrixArray) {
    const int n = hostMatrix->n, m = hostMatrix->m;
    Matrix* temp = new Matrix(n, m);

    const int matrix_size = sizeof(double) * n * m;
    CCE(hipMalloc(&temp->matrix, matrix_size));
    CCE(hipMemcpy(temp->matrix, hostMatrix->matrix, matrix_size, hipMemcpyHostToDevice));

    CCE(hipMalloc(deviceMatrix, sizeof(Matrix) * 1));
    CCE(hipMemcpy(*deviceMatrix, temp, sizeof(Matrix) * 1, hipMemcpyHostToDevice));

    *deviceMatrixArray = temp->matrix;
    temp->matrix = NULL;
    delete temp;
}

void copyMatrixFromDeviceToHost(double* deviceMatrixArray, Matrix** hostMatrix, int n, int m) {
    *hostMatrix = new Matrix(n, m);
    CCE(
        hipMemcpy(
            (*hostMatrix)->matrix,
            deviceMatrixArray,
            sizeof(double) * n * m,
            hipMemcpyDeviceToHost
        )
    );
}

/*
    QRDecompostion is part of SVD and should be called from host
    Matrix* a - pointer to matrix on host
    @return pair of Q and R matrix on host
*/
pair<Matrix*, Matrix*> QRDecompositionNaive(Matrix *a) {
    int n = a->n;
    int m = a->m;
    Matrix* Q = new Matrix(n, m);
    Matrix* R = new Matrix(m, m);
    for (int i = 0; i < m; i++) {
        auto ai = subMatrix(a, 0, n + 0, i + 0, i + 1);
        for (int k = 0; k < i; k++) {
            Matrix* qk = subMatrix(Q, 0, n + 0, k + 0, k + 1);
            Matrix* qkt = transpose(qk);
            Matrix* tempMultiply = multiply(qkt, ai);
            double v = -1 * tempMultiply->get(0, 0);
            Matrix* tmp = multiply(qk, v);
            Matrix* temp_ai = sum(ai, tmp);
            delete ai;
            ai = temp_ai;
            R->set(k, i, -1 * v);
            delete qk;
            delete qkt;
            delete tmp;
            delete tempMultiply;
        }
        R->set(i, i, vectorColLength(ai));
        auto nai = vectorColNormalize(ai);
        for (int k = 0; k < n; k++) {
            double nk0 = nai->get(k, 0);
            Q->set(k, i, nk0);
        }
        delete ai;
        delete nai;
    }
    return make_pair(Q, R);
}

Matrix* multiply_wrapper(Matrix* a, Matrix* b) {
    // Logic
    // 1. copy matrixes to device
    // 2. call multiply kernel
    // 3. copy results to host
    // 4. free allocated device memory

    Matrix* ab = new Matrix(a->n, b->m);

    // part 1 start
    Matrix *a_dev, *b_dev, *ab_dev;
    double *a_arr, *b_arr, *ab_arr;
    copyMatrixFromHostToDevice(a, &a_dev, &a_arr);
    copyMatrixFromHostToDevice(b, &b_dev, &b_arr);
    copyMatrixFromHostToDevice(ab, &ab_dev, &ab_arr);
    // part 1 end

    // part 2 start
    multiply<<<128, 32>>>(a_dev, b_dev, ab_dev);
    CCE(hipGetLastError())
    // part 2 end 

    // part 3 start
    copyMatrixFromDeviceToHost(ab_arr, &ab, a->n, b->m);
    // part 3 end 


    // part 4 start
    CCE(hipFree(a_arr));
    CCE(hipFree(a_dev));
    CCE(hipFree(b_arr));
    CCE(hipFree(b_dev));
    CCE(hipFree(ab_arr));
    CCE(hipFree(ab_dev));
    // part 4 end
    return ab;
}

Triple* SVDDecomposition(Matrix *a, int rank, double eps) {
    int n = a->n;
    int m = a->m;
    auto u = randomMatrix(n, rank), sgm = randomMatrix(rank, rank), v = randomMatrix(m, rank);
    auto at = transpose(a);
    double err = 1e9;
    for (; err > eps;) {

        // auto av = multiply(a, v);
        auto av = multiply_wrapper(a, v);

        // show(av, a->n, rank);
        // show(av_test, a->n, rank);
        // exit(0);
        auto qr_av = QRDecompositionNaive(av);

        Matrix* u_tmp = subMatrix(qr_av.first, 0, n, 0, rank);
        delete u;
        u = u_tmp;

        auto atu = multiply(at, u);
        auto qr_atu = QRDecompositionNaive(atu);

        Matrix* v_tmp = subMatrix(qr_atu.first, 0, m, 0, rank);
        delete v;
        v = v_tmp;

        Matrix* sgm_tmp = subMatrix(qr_atu.second, 0, rank, 0, rank);
        delete sgm;
        sgm = sgm_tmp;

        // find error e = || A*V - U*SGM||
        // av = multiply(a, v);
        auto usgm = multiply(u, sgm);
        double revert = -1;
        Matrix* usgmt = multiply(usgm, revert);
        auto difff = sum(av, usgmt);
        err = matrixNorm(difff);
        // double av_diff = diff(av, av_test);
        printf("Iteration ended, error=%f, diff=%f\n", err, 0.f);

        delete av;
        // delete av_test;
        delete qr_av.first;
        delete qr_av.second;
        delete atu;
        delete qr_atu.first;
        delete qr_atu.second;
        delete usgm;
        delete usgmt;
        delete difff;
    }
    delete at;
    return new Triple(u, sgm, v);
}